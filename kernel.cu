#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <ctime>


// Helper function for using CUDA to call kernel functions
hipError_t cuda_code(float* a, float* x, int N, int dim);
__device__ float sum = 0;
__global__ void elimination(float *a, int N, int m);
__global__ void substitution(int i, int N, float *row, float *a, float*x);
__global__ void kernel_func(float* a, float* x, int N);
void readm(float* m, int n);
void printm(float *m, float*x, int n);

int main() {
	float* m;
	float* x;
	int dim;
	int N = 1000;
	dim = (N + 1) * N;

	m = (float*)malloc(dim * sizeof(float));
	x = (float*)malloc(N * sizeof(float));

	readm(m, N);
	clock_t begin = clock();
	hipError_t cudaStatus = cuda_code(m, x, N, dim);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	clock_t end = clock();
	double elapsed_secs = double(end - begin);// / CLOCKS_PER_SEC;
	printm(m, x, N);
	std::cout << "Elapsed time: " << elapsed_secs << std::endl;
	return 0;
}

hipError_t cuda_code(float* a, float* x, int N, int dim)
{
	hipError_t cudaStatus;
	size_t size1 = dim * sizeof(float);
	size_t size2 = N * sizeof(float);
	float* dev_a;
	float* dev_x;

	// Allocate GPU buffers for two vectors (one input, one output).
	cudaStatus = hipMalloc((void**)&dev_a, size1);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_x, size2);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, dim * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	// Launch a kernel on the GPU with one thread for each element.
	kernel_func << <1, 1 >> > (dev_a, dev_x, N);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(a, dev_a, size1, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipMemcpy(x, dev_x, size2, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	cudaStatus = hipFree(dev_a);
	cudaStatus = hipFree(dev_x);

	return cudaStatus;
}
__global__ void elimination(float *a, int N, int m)
{
	int i = m + threadIdx.x + blockIdx.x * blockDim.x;
	int j = m + blockIdx.y * blockDim.y + threadIdx.y;
	int ij;		//element i,j of the matrix
	int ebmri;	//element below m, row i
	int mm;		//pivot previous row, diagonal element
	int prcj;	//element of previous row at column j

	if (i > m && i < N)
	{
		if (j > m && j < (N + 1))
		{
			ij = j + (N + 1)*i;
			ebmri = m + (N + 1)*i;
			mm = m + (N + 1)*m;
			prcj = j + (N + 1)*m;
			a[ij] = a[ij] - ((a[ebmri] / a[mm])*a[prcj]);
		}
	}
	__syncthreads();
}
__global__ void substitution(int i, int N, float *row, float *a, float*x) {
	int j = i + blockIdx.x * blockDim.x + threadIdx.x;
	int ij;		//element i,j of the matrix
	if (j > i && j < N)
	{
		ij = j + (N + 1)*i;
		row[j] = a[ij] * x[j];
		atomicAdd(&sum, row[j]);
	}
	__syncthreads();
}
__global__ void kernel_func(float* a, float* x, int N)
{
	//dim3 Blocks(32, 32);	//Number of blocks per axis
	//dim3 Blocksize(32, 32);	//Number of threads per Block per axis
	//elimination
	for (int m = 0; m < N; m++)
	{
		int Dx = ceilf((float)(N - m) / 16.0);
		int Dy = ceilf((float)(N + 1 - m) / 16.0);
		dim3 Blocks(Dx, Dy);
		dim3 Blocksize(16, 16);
		elimination << <Blocks, Blocksize >> > (a, N, m);
		hipDeviceSynchronize();
	}
	//The last element of x, can be solved directly as follows
	x[N - 1] = a[N*(N + 1) - 1] / a[N*(N + 1) - 2];
	//This array stores temporarily at location i, the multiplication a[ij] * x[j]
	//In order to add each of of them atomically into "sum"
	float *row;
	row = (float*)malloc((N + 1) * sizeof(float));
	//backwards substitution
	int eltb;
	int eltij;
	for (int i = N - 2; 0 <= i; i--)
	{
		sum = 0;
		int Dy = ceilf((float)(N + 1 - i) / 32.0);

		substitution << <Dy, 32 >> > (i, N, row, a, x);
		hipDeviceSynchronize();
		eltb = N + (N + 1)*i;
		eltij = i + (N + 1)*i;
		x[i] = (a[eltb] - sum) / a[eltij];
	}
	printf("Hello");
}
void readm(float* m, int n) {
	using namespace std;
	ifstream myfile;
	myfile.open("matrix.txt");

	if (myfile.is_open()) {
		for (int j = 0; j < n; j++)
		{
			for (int i = 0; i < (n + 1); i++)
			{
				int ij = i + ((n + 1) * j);
				myfile >> m[ij];
			}
		}

	}


	myfile.close();
}
void printm(float* m, float* x, int n) {
	std::ofstream myfile;
	myfile.open("matrix2.txt");

	if (myfile.is_open()) {
		for (int j = 0; j < n; j++)
		{
			for (int i = 0; i < n + 1; i++)
			{
				myfile << m[i + (n + 1) * j] << "\t";
			}
			myfile << "\n";
		}
		for (int j = 0; j < n; j++)
		{
			myfile << x[j] << "\n";
		}

	}
	myfile.close();
}